#include "hip/hip_runtime.h"
//
//  Peaks.cu
//
//
//  Created by Hemant Sharma on 2015/07/04.
//

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <sys/stat.h>
#include <string.h>
#include <ctype.h>
#include <stdint.h>
#include <stdbool.h>
#include <sys/types.h>
#include <errno.h>
#include <stdarg.h>
#include <fcntl.h>
#include "nldrmd.cuh"

#define deg2rad 0.0174532925199433
#define rad2deg 57.2957795130823
#define MAXNHKLS 5000
#define CalcNorm3(x,y,z) sqrt((x)*(x) + (y)*(y) + (z)*(z))
#define CalcNorm2(x,y) sqrt((x)*(x) + (y)*(y))
typedef uint16_t pixelvalue;

static inline
pixelvalue**
allocMatrixPX(int nrows, int ncols)
{
    pixelvalue** arr;
    int i;
    arr = (pixelvalue **) malloc(nrows * sizeof(*arr));
    if (arr == NULL ) {
        return NULL;
    }
    for ( i = 0 ; i < nrows ; i++) {
        arr[i] = (pixelvalue*) malloc(ncols * sizeof(*arr[i]));
        if (arr[i] == NULL ) {
            return NULL;
        }
    }
    return arr;
}

static inline
void
FreeMemMatrixPx(pixelvalue **mat,int nrows)
{
    int r;
    for ( r = 0 ; r < nrows ; r++) {
        free(mat[r]);
    }
    free(mat);
}

static inline double sind(double x){return sin(deg2rad*x);}
static inline double cosd(double x){return cos(deg2rad*x);}
static inline double tand(double x){return tan(deg2rad*x);}
static inline double asind(double x){return rad2deg*(asin(x));}
static inline double acosd(double x){return rad2deg*(acos(x));}
static inline double atand(double x){return rad2deg*(atan(x));}

static inline int CheckDirectoryCreation(char Folder[1024])
{
	int e;
    struct stat sb;
	char totOutDir[1024];
	sprintf(totOutDir,"%s/",Folder);
    e = stat(totOutDir,&sb);
    if (e!=0 && errno == ENOENT){
		printf("Output directory did not exist, creating %s\n",totOutDir);
		e = mkdir(totOutDir,S_IRWXU);
		if (e !=0) {printf("Could not make the directory. Exiting\n");return 0;}
	}
	return 1;
}

static inline void DoImageTransformations(int NrTransOpt, int TransOpt[10], pixelvalue *Image, int NrPixels)
{
	int i,j,k,l,m;
    pixelvalue **ImageTemp1, **ImageTemp2;
    ImageTemp1 = allocMatrixPX(NrPixels,NrPixels);
    ImageTemp2 = allocMatrixPX(NrPixels,NrPixels);
	for (k=0;k<NrPixels;k++) for (l=0;l<NrPixels;l++) ImageTemp1[k][l] = Image[(NrPixels*k)+l];
	for (k=0;k<NrTransOpt;k++) {
		if (TransOpt[k] == 1){
			for (l=0;l<NrPixels;l++) for (m=0;m<NrPixels;m++) ImageTemp2[l][m] = ImageTemp1[l][NrPixels-m-1]; //Inverting Y.
		} else if (TransOpt[k] == 2){
			for (l=0;l<NrPixels;l++) for (m=0;m<NrPixels;m++) ImageTemp2[l][m] = ImageTemp1[NrPixels-l-1][m]; //Inverting Z.
		} else if (TransOpt[k] == 3){
			for (l=0;l<NrPixels;l++) for (m=0;m<NrPixels;m++) ImageTemp2[l][m] = ImageTemp1[m][l];
		} else if (TransOpt[k] == 0){
			for (l=0;l<NrPixels;l++) for (m=0;m<NrPixels;m++) ImageTemp2[l][m] = ImageTemp1[l][m];
		}
		for (l=0;l<NrPixels;l++) for (m=0;m<NrPixels;m++) ImageTemp1[l][m] = ImageTemp2[l][m];
	}
	for (k=0;k<NrPixels;k++) for (l=0;l<NrPixels;l++) Image[(NrPixels*k)+l] = ImageTemp2[k][l];
	FreeMemMatrixPx(ImageTemp1,NrPixels);
	FreeMemMatrixPx(ImageTemp2,NrPixels);
}

static void
check (int test, const char * message, ...)
{
    if (test) {
        va_list args;
        va_start (args, message);
        vfprintf (stderr, message, args);
        va_end (args);
        fprintf (stderr, "\n");
        exit (EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]){ // Arguments: parameter file name, file number, ring number
	clock_t start, end;
	if (argc != 3){
		printf("Not enough arguments, exiting. Use as:\n\t\t\n");
		return 1;
	}
    double diftotal;
    start = clock();
    // Read params file.
    char *ParamFN;
    FILE *fileParam;
    ParamFN = argv[1];
    char aline[1000];
    fflush(stdout);
    fileParam = fopen(ParamFN,"r");
    if (fileParam == NULL){
		printf("Parameter file could not be read. Exiting\n");
		return 1;
	}
    check (fileParam == NULL,"%s file not found: %s", ParamFN, strerror(errno));
    char *str, dummy[1000], Folder[1024], FileStem[1024], *TmpFolder, darkcurrentfilename[1024], floodfilename[1024], Ext[1024],RawFolder[1024];
    TmpFolder = "Temp";
    int LowNr,FileNr;
    FileNr = atoi(argv[2]);
    double Thresh, bc=1, Ycen, Zcen, IntSat, OmegaStep, OmegaFirstFile, Lsd, px, Width, Wavelength, LatticeConstant,MaxRingRad;
    int CellStruct,NrPixels,Padding = 6, StartNr;
    char fs[1024];
    int LayerNr;
    int NrTransOpt=0;
    int TransOpt[10];
    int StartFileNr, NrFilesPerSweep;
    int DoFullImage = 0;
    int FrameNrOmeChange = 1;
    double OmegaMissing = 0, MisDir;
    while (fgets(aline,1000,fileParam)!=NULL){
		printf("%s\n",aline);
		fflush(stdout);
        str = "StartFileNr ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &StartFileNr);
            continue;
        }
        str = "DoFullImage ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &DoFullImage);
            continue;
        }
        str = "NrFilesPerSweep ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &NrFilesPerSweep);
            continue;
        }
        str = "Ext ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %s", dummy, Ext);
            continue;
        }
        str = "RawFolder ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %s", dummy, RawFolder);
            continue;
        }
        str = "Folder ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %s", dummy, Folder);
            continue;
        }
        str = "FileStem ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %s", dummy, fs);
            continue;
        }
        str = "Dark ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %s", dummy, darkcurrentfilename);
            continue;
        }
        str = "Flood ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %s", dummy, floodfilename);
            continue;
        }
        str = "LowerBoundThreshold ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &Thresh);
            continue;
        }
        str = "BeamCurrent ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &bc);
            continue;
        }
        str = "BC ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf %lf", dummy, &Ycen, &Zcen);
            continue;
        }
        str = "UpperBoundThreshold ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &IntSat);
            continue;
        }
        str = "OmegaStep ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &OmegaStep);
            continue;
        }
        str = "OmegaFirstFile ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &OmegaFirstFile);
            continue;
        }
        str = "px ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &px);
            continue;
        }
        str = "Width ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &Width);
            continue;
        }
        str = "LayerNr ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &LayerNr);
            continue;
        }
        str = "CellStruct ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &CellStruct);
            continue;
        }
        str = "NrPixels ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &NrPixels);
            continue;
        }
        str = "Padding ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &Padding);
            continue;
        }
        str = "Wavelength ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &Wavelength);
            continue;
        }
        str = "Lsd ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &Lsd);
            continue;
        }
        str = "StartNr ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &StartNr);
            continue;
        }
        str = "MaxRingRad ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %lf", dummy, &MaxRingRad);
            continue;
        }
        str = "ImTransOpt ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d", dummy, &TransOpt[NrTransOpt]);
            NrTransOpt++;
            continue;
        }
        str = "FrameOmeChange ";
        LowNr = strncmp(aline,str,strlen(str));
        if (LowNr==0){
            sscanf(aline,"%s %d %lf %lf", dummy, &FrameNrOmeChange, &OmegaMissing, &MisDir);
            continue;
        }
	}
	printf("%f\n",Thresh);
	Width = Width/px;
	int i,j,k;
    for (i=0;i<NrTransOpt;i++){
        if (TransOpt[i] < 0 || TransOpt[i] > 3){printf("TransformationOptions can only be 0, 1, 2 or 3.\nExiting.\n");return 0;}
        printf("TransformationOptions: %d ",TransOpt[i]);
        if (TransOpt[i] == 0) printf("No change.\n");
        else if (TransOpt[i] == 1) printf("Flip Left Right.\n");
        else if (TransOpt[i] == 2) printf("Flip Top Bottom.\n");
        else printf("Transpose.\n");
    }

}
